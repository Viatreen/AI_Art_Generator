// Standard Library
#include <iostream>

#include <hip/driver_types.h>

// File header
#include "AI_Art_Generator/Utils/error_check_cuda.hpp"

static std::string cuda_return_error_string(hipError_t error)
{
    switch (error)
    {
    case hipErrorInvalidValue:                   return "hipErrorInvalidValue";
    case hipErrorOutOfMemory:               return "hipErrorOutOfMemory";
    case hipErrorNotInitialized:            return "hipErrorNotInitialized";
    case hipErrorDeinitialized:                return "hipErrorDeinitialized";
    case hipErrorProfilerDisabled:               return "hipErrorProfilerDisabled";
    case hipErrorProfilerNotInitialized:         return "hipErrorProfilerNotInitialized";
    case hipErrorProfilerAlreadyStarted:         return "hipErrorProfilerAlreadyStarted";
    case hipErrorProfilerAlreadyStopped:         return "hipErrorProfilerAlreadyStopped";
    case hipErrorInvalidConfiguration:           return "hipErrorInvalidConfiguration";
    case hipErrorInvalidPitchValue:              return "hipErrorInvalidPitchValue";
    case hipErrorInvalidSymbol:                  return "hipErrorInvalidSymbol";
    case cudaErrorInvalidHostPointer:             return "cudaErrorInvalidHostPointer";
    case hipErrorInvalidDevicePointer:           return "hipErrorInvalidDevicePointer";
    case hipErrorInvalidTexture:                 return "hipErrorInvalidTexture";
    case cudaErrorInvalidTextureBinding:          return "cudaErrorInvalidTextureBinding";
    case hipErrorInvalidChannelDescriptor:       return "hipErrorInvalidChannelDescriptor";
    case hipErrorInvalidMemcpyDirection:         return "hipErrorInvalidMemcpyDirection";
    case cudaErrorAddressOfConstant:              return "cudaErrorAddressOfConstant";
    case cudaErrorTextureFetchFailed:             return "cudaErrorTextureFetchFailed";
    case cudaErrorTextureNotBound:                return "cudaErrorTextureNotBound";
    case cudaErrorSynchronizationError:           return "cudaErrorSynchronizationError";
    case cudaErrorInvalidFilterSetting:           return "cudaErrorInvalidFilterSetting";
    case cudaErrorInvalidNormSetting:             return "cudaErrorInvalidNormSetting";
    case cudaErrorMixedDeviceExecution:           return "cudaErrorMixedDeviceExecution";
    case cudaErrorNotYetImplemented:              return "cudaErrorNotYetImplemented";
    case cudaErrorMemoryValueTooLarge:            return "cudaErrorMemoryValueTooLarge";
    case cudaErrorStubLibrary:                    return "cudaErrorStubLibrary";
    case hipErrorInsufficientDriver:             return "hipErrorInsufficientDriver";
    case cudaErrorCallRequiresNewerDriver:        return "cudaErrorCallRequiresNewerDriver";
    case cudaErrorInvalidSurface:                 return "cudaErrorInvalidSurface";
    case cudaErrorDuplicateVariableName:          return "cudaErrorDuplicateVariableName";
    case cudaErrorDuplicateTextureName:           return "cudaErrorDuplicateTextureName";
    case cudaErrorDuplicateSurfaceName:           return "cudaErrorDuplicateSurfaceName";
    case cudaErrorDevicesUnavailable:             return "cudaErrorDevicesUnavailable";
    case cudaErrorIncompatibleDriverContext:      return "cudaErrorIncompatibleDriverContext";
    case hipErrorMissingConfiguration:           return "hipErrorMissingConfiguration";
    case hipErrorPriorLaunchFailure:             return "hipErrorPriorLaunchFailure";
    case cudaErrorLaunchMaxDepthExceeded:         return "cudaErrorLaunchMaxDepthExceeded";
    case cudaErrorLaunchFileScopedTex:            return "cudaErrorLaunchFileScopedTex";
    case cudaErrorLaunchFileScopedSurf:           return "cudaErrorLaunchFileScopedSurf";
    case cudaErrorSyncDepthExceeded:              return "cudaErrorSyncDepthExceeded";
    case cudaErrorLaunchPendingCountExceeded:     return "cudaErrorLaunchPendingCountExceeded";
    case hipErrorInvalidDeviceFunction:          return "hipErrorInvalidDeviceFunction";
    case hipErrorNoDevice:                       return "hipErrorNoDevice";
    case hipErrorInvalidDevice:                  return "hipErrorInvalidDevice";
    case cudaErrorDeviceNotLicensed:              return "cudaErrorDeviceNotLicensed";
    case cudaErrorSoftwareValidityNotEstablished: return "cudaErrorSoftwareValidityNotEstablished";
    case cudaErrorStartupFailure:                 return "cudaErrorStartupFailure";
    case hipErrorInvalidImage:             return "hipErrorInvalidImage";
    case hipErrorInvalidContext:            return "hipErrorInvalidContext";
    case hipErrorMapFailed:          return "hipErrorMapFailed";
    case hipErrorUnmapFailed:        return "hipErrorUnmapFailed";
    case hipErrorArrayIsMapped:                  return "hipErrorArrayIsMapped";
    case hipErrorAlreadyMapped:                  return "hipErrorAlreadyMapped";
    case hipErrorNoBinaryForGpu:         return "hipErrorNoBinaryForGpu";
    case hipErrorAlreadyAcquired:                return "hipErrorAlreadyAcquired";
    case hipErrorNotMapped:                      return "hipErrorNotMapped";
    case hipErrorNotMappedAsArray:               return "hipErrorNotMappedAsArray";
    case hipErrorNotMappedAsPointer:             return "hipErrorNotMappedAsPointer";
    case hipErrorECCNotCorrectable:               return "hipErrorECCNotCorrectable";
    case hipErrorUnsupportedLimit:               return "hipErrorUnsupportedLimit";
    case hipErrorContextAlreadyInUse:             return "hipErrorContextAlreadyInUse";
    case hipErrorPeerAccessUnsupported:          return "hipErrorPeerAccessUnsupported";
    case hipErrorInvalidKernelFile:                     return "hipErrorInvalidKernelFile";
    case hipErrorInvalidGraphicsContext:         return "hipErrorInvalidGraphicsContext";
    case cudaErrorNvlinkUncorrectable:            return "cudaErrorNvlinkUncorrectable";
    case cudaErrorJitCompilerNotFound:            return "cudaErrorJitCompilerNotFound";
    case cudaErrorUnsupportedPtxVersion:          return "cudaErrorUnsupportedPtxVersion";
    case cudaErrorJitCompilationDisabled:         return "cudaErrorJitCompilationDisabled";
    case cudaErrorUnsupportedExecAffinity:        return "cudaErrorUnsupportedExecAffinity";
    case hipErrorInvalidSource:                  return "hipErrorInvalidSource";
    case hipErrorFileNotFound:                   return "hipErrorFileNotFound";
    case hipErrorSharedObjectSymbolNotFound:     return "hipErrorSharedObjectSymbolNotFound";
    case hipErrorSharedObjectInitFailed:         return "hipErrorSharedObjectInitFailed";
    case hipErrorOperatingSystem:                return "hipErrorOperatingSystem";
    case hipErrorInvalidHandle:          return "hipErrorInvalidHandle";
    case hipErrorIllegalState:                   return "hipErrorIllegalState";
    case hipErrorNotFound:                 return "hipErrorNotFound";
    case hipErrorNotReady:                       return "hipErrorNotReady";
    case hipErrorIllegalAddress:                 return "hipErrorIllegalAddress";
    case hipErrorLaunchOutOfResources:           return "hipErrorLaunchOutOfResources";
    case hipErrorLaunchTimeOut:                  return "hipErrorLaunchTimeOut";
    case cudaErrorLaunchIncompatibleTexturing:    return "cudaErrorLaunchIncompatibleTexturing";
    case hipErrorPeerAccessAlreadyEnabled:       return "hipErrorPeerAccessAlreadyEnabled";
    case hipErrorPeerAccessNotEnabled:           return "hipErrorPeerAccessNotEnabled";
    case hipErrorSetOnActiveProcess:             return "hipErrorSetOnActiveProcess";
    case hipErrorContextIsDestroyed:             return "hipErrorContextIsDestroyed";
    case hipErrorAssert:                         return "hipErrorAssert";
    case cudaErrorTooManyPeers:                   return "cudaErrorTooManyPeers";
    case hipErrorHostMemoryAlreadyRegistered:    return "hipErrorHostMemoryAlreadyRegistered";
    case hipErrorHostMemoryNotRegistered:        return "hipErrorHostMemoryNotRegistered";
    case cudaErrorHardwareStackError:             return "cudaErrorHardwareStackError";
    case cudaErrorIllegalInstruction:             return "cudaErrorIllegalInstruction";
    case cudaErrorMisalignedAddress:              return "cudaErrorMisalignedAddress";
    case cudaErrorInvalidAddressSpace:            return "cudaErrorInvalidAddressSpace";
    case cudaErrorInvalidPc:                      return "cudaErrorInvalidPc";
    case hipErrorLaunchFailure:                  return "hipErrorLaunchFailure";
    case hipErrorCooperativeLaunchTooLarge:      return "hipErrorCooperativeLaunchTooLarge";
    case cudaErrorNotPermitted:                   return "cudaErrorNotPermitted";
    case hipErrorNotSupported:                   return "hipErrorNotSupported";
    case cudaErrorSystemNotReady:                 return "cudaErrorSystemNotReady";
    case cudaErrorSystemDriverMismatch:           return "cudaErrorSystemDriverMismatch";
    case cudaErrorCompatNotSupportedOnDevice:     return "cudaErrorCompatNotSupportedOnDevice";
    case cudaErrorMpsConnectionFailed:            return "cudaErrorMpsConnectionFailed";
    case cudaErrorMpsRpcFailure:                  return "cudaErrorMpsRpcFailure";
    case cudaErrorMpsServerNotReady:              return "cudaErrorMpsServerNotReady";
    case cudaErrorMpsMaxClientsReached:           return "cudaErrorMpsMaxClientsReached";
    case cudaErrorMpsMaxConnectionsReached:       return "cudaErrorMpsMaxConnectionsReached";
    case cudaErrorMpsClientTerminated:            return "cudaErrorMpsClientTerminated";
    case cudaErrorCdpNotSupported:                return "cudaErrorCdpNotSupported";
    case cudaErrorCdpVersionMismatch:             return "cudaErrorCdpVersionMismatch";
    case hipErrorStreamCaptureUnsupported:       return "hipErrorStreamCaptureUnsupported";
    case hipErrorStreamCaptureInvalidated:       return "hipErrorStreamCaptureInvalidated";
    case hipErrorStreamCaptureMerge:             return "hipErrorStreamCaptureMerge";
    case hipErrorStreamCaptureUnmatched:         return "hipErrorStreamCaptureUnmatched";
    case hipErrorStreamCaptureUnjoined:          return "hipErrorStreamCaptureUnjoined";
    case hipErrorStreamCaptureIsolation:         return "hipErrorStreamCaptureIsolation";
    case hipErrorStreamCaptureImplicit:          return "hipErrorStreamCaptureImplicit";
    case hipErrorCapturedEvent:                  return "hipErrorCapturedEvent";
    case hipErrorStreamCaptureWrongThread:       return "hipErrorStreamCaptureWrongThread";
    case cudaErrorTimeout:                        return "cudaErrorTimeout";
    case hipErrorGraphExecUpdateFailure:         return "hipErrorGraphExecUpdateFailure";
    case cudaErrorExternalDevice:                 return "cudaErrorExternalDevice";
    case cudaErrorInvalidClusterSize:             return "cudaErrorInvalidClusterSize";
    case hipErrorUnknown:                        return "hipErrorUnknown";
    case cudaErrorApiFailureBase:                 return "cudaErrorApiFailureBase";
    }

    return "<unknown>";
}

void cuda_check_expanded(hipError_t result, const char *function_name, const char *filename, int line_number)
{
    if(result)
    {
        std::cout << "CUDA Error: " << cuda_return_error_string(result) << " (" << result << "), Function name: " << function_name << ", Filename:" << filename << ":" << line_number << std::endl;
    }

    return;
}
