// Standard Library
#include <iostream>

// Project Headers
#include "AI_Art_Generator/Utils/gpu_info_print.hpp"
#include "AI_Art_Generator/Utils/config.hpp"
#include "AI_Art_Generator/Utils/cuda_helper.hpp"
#include "AI_Art_Generator/Utils/error_check_cuda.hpp"
#include "AI_Art_Generator/Utils/file_writer_neural_net.hpp"
#include "MNIST/MNIST_neural_network.hpp"

int main()
{
    std::cout << "MNIST Begin" << std::endl;

    int status = util::gpu_info_print();
    if (status) {
        return status; // No need for print statement. Print comes from within gpu_info_print
    }

    neural_network::NN<nn_float> *nn;

    hipMalloc(&nn, sizeof(neural_network::NN<nn_float>));

    neural_network::populate<<<num_blocks(INPUT_SIZE), BLOCK_SIZE>>>(nn);
    cuda_check_sync;
 
    void *kernel_args[] = { (void*)&nn };
    hipLaunchCooperativeKernel((void*)neural_network::run, num_blocks(FIRST_LAYER_OUTPUT_SIZE), BLOCK_SIZE, kernel_args);
    cuda_check_sync;

    util::write_2_csv(nn);

    return 0;
}
